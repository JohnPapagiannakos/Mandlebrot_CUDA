#include "hip/hip_runtime.h"
// Author: Yannis Papagiannakos
#define USE_CUDA 1

#include <string>
#include <iterator>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <array>
#include <cmath>
#include <complex>

#include <chrono>
#include <ctime>
#include <unistd.h>

#include "masterlib.hpp"

#include "OpenGL/plot.hpp"

int main ( void ){
    using namespace std::complex_literals;

    // const size_t dim = 1000;
    
    // std::array<size_t, 2> Dims = {dim, dim};
    std::array<size_t, 2> Dims = {1920, 1080};
    size_t prod_dims = Dims[0] * Dims[1];

    const int MAX_ITERS = 500;

    const int MAX_WHILE_ITERS = 10;

    double ratio = Dims[0] / Dims[1];

    double offset_x = ratio;
    double offset_y = 1.0;

    std::array<double, 2> center = {0, 0};

    // double alpha = 3*M_PI_4; // pi/4
    // DoubleComplex tmp_const_c = 0.7885 * std::exp(1i * alpha);
    
    // DoubleComplex tmp_const_c = -1.476;

    // DoubleComplex tmp_const_c = -0.79 + 0.15i;

    DoubleComplex tmp_const_c = 0.28 + 0.008i;

    std::cout << "c=" << real(tmp_const_c);
    if(imag(tmp_const_c)>=0)
        std::cout << "+" << imag(tmp_const_c) << "i" << std::endl;
    else
        std::cout << imag(tmp_const_c) << "i" << std::endl;
    hipDoubleComplex const_c;
    const_c.x = real(tmp_const_c);
    const_c.y = imag(tmp_const_c);

    std::array<double, 2> XLIM = {center[0] - offset_x, center[0] + offset_x};
    std::array<double, 2> YLIM = {center[1] - offset_y, center[1] + offset_y};

    // Create meshgrid
    hipDoubleComplex *z0;
    hipMallocManaged((void **)&z0, Dims[0] * Dims[1] * sizeof(hipDoubleComplex));
    cudameshgrid(XLIM, YLIM, Dims, z0);

    double *count;
    hipMallocManaged((void **)&count, prod_dims * sizeof(double)); // unified mem.

    std::fill(&count[0], &count[prod_dims - 1], 1.0);
    
    // Start timers
    hipDeviceSynchronize();
    v2::cuJuliaOp2(z0, const_c, count, prod_dims, MAX_ITERS);
    // v2::cuJuliaOp3(z0, const_c, count, prod_dims, MAX_ITERS);
    hipDeviceSynchronize();


    std::vector<double> _data(prod_dims, 1);
    for (size_t idx = 0; idx < prod_dims; idx++)
    {
        _data[idx] = count[idx];
    }

    // Illustrate fractal
    figure<double> fig(Dims);
    fig.newFigure("Mandelbrot Set");
    fig.plotRGB(_data);
    // fig.showFigure();
    // sleep(1);
    std::chrono::time_point<std::chrono::system_clock> start, end;

    int while_iters = 0;
    while(while_iters < MAX_WHILE_ITERS)
    {
        start = std::chrono::system_clock::now();

        offset_x *= 0.8;
        offset_y *= 0.8;
        center[0] = offset_x;
        center[1] = offset_y;
        XLIM = {center[0] - offset_x, center[0] + offset_x};
        YLIM = {center[1] - offset_y, center[1] + offset_y};

        // Create meshgrid
        cudameshgrid(XLIM, YLIM, Dims, z0);

        std::fill(&count[0], &count[prod_dims - 1], 1.0);
       
        // Start timers
        hipDeviceSynchronize();
        v2::cuJuliaOp2(z0, const_c, count, prod_dims, MAX_ITERS);
        // v2::cuJuliaOp3(z0, const_c, count, prod_dims, MAX_ITERS);
        hipDeviceSynchronize();


        std::vector<double> _data(prod_dims, 1);
        for (size_t idx = 0; idx < prod_dims; idx++)
        {
            _data[idx] = count[idx];
        }

        fig.plotRGB(_data);
        // sleep(1);
        end = std::chrono::system_clock::now();

        std::chrono::duration<double> elapsed_seconds = end - start;

        std::cout << "elapsed time: " << elapsed_seconds.count() << "s\n";
        while_iters++;
    }

    hipFree(z0);
    hipFree(count);

    return 0;
}
