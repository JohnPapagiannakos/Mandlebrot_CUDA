#include "hip/hip_runtime.h"
// Author: Yannis Papagiannakos

#include "cudaoperators.hpp"

__device__ inline double margind(double x, double y)
{
    return sqrt((x * x) + (y * y));
}

__global__ void addv(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

__global__ void juliaOp2v(hipDoubleComplex *a, const hipDoubleComplex c, double *count, int n)
{
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    double margin;

    int bool_count;

    // Make sure we do not go out of bounds
    if (id < n)
    {
        double a_x = c.x;
        double a_y = c.y;

        a_x += a[id].x * a[id].x - a[id].y * a[id].y;
        a_y += 2*a[id].x * a[id].y;
        margin = margind(a_x, a_y);
        bool_count = (margin <= 2);
        count[id] = count[id] + bool_count;   
        
        a[id].x = a_x;
        a[id].y = a_y;
    }
}

__global__ void juliaOp3v(hipDoubleComplex *a, const hipDoubleComplex c, double *count, int n)
{
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    double margin;

    int bool_count;

    // Make sure we do not go out of bounds
    if (id < n)
    {
        double a_x = c.x;
        double a_y = c.y;

        a_x += a[id].x * (a[id].x * a[id].x - 3*a[id].y * a[id].y);
        a_y += a[id].y * (3*a[id].x * a[id].x - a[id].y * a[id].y);
        margin = margind(a_x, a_y);
        bool_count = (margin <= 2);
        count[id] = count[id] + bool_count;   
        
        a[id].x = a_x;
        a[id].y = a_y;
    }
}

__global__ void logv(double *a, int n)
{
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n)
        a[id] = log(a[id]);
}


extern "C" void cuaddv(double *A, double *B, double *C, int length)
{
    int blockSize, gridSize;
    blockSize = CUDA_VEC_BLOCK_SIZE;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)length / blockSize);

    addv<<<gridSize, blockSize>>>(A, B, C, length);
}   


extern "C" void cuJuliaOp2(hipDoubleComplex *z, const hipDoubleComplex c, double *count, int length, const int MAX_ITERS)
{
    int blockSize, gridSize;
    blockSize = CUDA_VEC_BLOCK_SIZE;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)length / blockSize);

    for(int iter=0; iter<=MAX_ITERS; iter++)
    {
        juliaOp2v<<<gridSize, blockSize>>>(z, c, count, length);
    }
    // hipDeviceSynchronize();
    logv<<<gridSize, blockSize>>>(count, length);
} 

extern "C" void cuJuliaOp3(hipDoubleComplex *z, const hipDoubleComplex c, double *count, int length, const int MAX_ITERS)
{
    int blockSize, gridSize;
    blockSize = CUDA_VEC_BLOCK_SIZE;

    // Number of thread blocks in grid
    gridSize = (int)ceil((float)length / blockSize);

    for(int iter=0; iter<=MAX_ITERS; iter++)
    {
        juliaOp3v<<<gridSize, blockSize>>>(z, c, count, length);
    }
    // hipDeviceSynchronize();
    logv<<<gridSize, blockSize>>>(count, length);
} 